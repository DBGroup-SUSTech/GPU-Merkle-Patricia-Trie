// #define GETH 1
#define LEDGERDB 1

#include <unistd.h>

#include <mutex>
#include <tuple>

#include "libgmpt.h"
#include "mpt/cpu_mpt.cuh"
#include "mpt/gpu_mpt.cuh"
#include "mpt/node.cuh"
#include "util/experiments.cuh"
#include "util/utils.cuh"

struct Tries {
  GpuMPT::Compress::MPT *state_trie;
  GpuMPT::Compress::MPT *transaction_trie;
  GpuMPT::Compress::MPT *receipt_trie;
};

static struct Tries *tries = nullptr;

struct ThreadSafeMPT {
  GpuMPT::Compress::MPT mpt;
  std::mutex mu;
  uint8_t hash[HASH_SIZE];
  int hash_size;
};

// TODO: Redefine ALLOCATOR
int preprocess() {
  printf("preprocess()\n");
  CHECK_ERROR(hipSetDevice(1));
  CHECK_ERROR(hipDeviceReset());
  if (tries != nullptr) {
    // delete (GpuMPT::Compress::MPT *)tries->state_trie;
    // delete (GpuMPT::Compress::MPT *)tries->receipt_trie;
    // delete (GpuMPT::Compress::MPT *)tries->transaction_trie;
    delete tries;
  }
  GPUHashMultiThread::load_constants();
  tries = new Tries{};
  tries->state_trie = new GpuMPT::Compress::MPT{};
  tries->receipt_trie = new GpuMPT::Compress::MPT{};
  tries->transaction_trie = new GpuMPT::Compress::MPT{};
  printf("state_trie %p, receipt_trie %p, transaction_trie %p\n",
         tries->state_trie, tries->receipt_trie, tries->transaction_trie);
  return 1;
}

const uint8_t *build_mpt_2phase(enum TrieType trie_type,
                                const uint8_t *keys_hexs, int *keys_hexs_indexs,
                                const uint8_t *values_bytes,
                                int64_t *values_bytes_indexs,
                                const uint8_t **values_hps, int insert_num) {
  CHECK_ERROR(hipSetDevice(1));
  assert(tries != nullptr);
  GpuMPT::Compress::MPT *mpt = nullptr;
  if (trie_type == TrieType::RECEIPT_TRIE) {
    mpt = tries->receipt_trie;
  } else if (trie_type == TrieType::STATE_TRIE) {
    mpt = tries->state_trie;
  } else if (trie_type == TrieType::TRANSACTION_TRIE) {
    mpt = tries->transaction_trie;
  }
  assert(mpt != nullptr);

  // choose the second GPU
  perf::CpuMultiTimer<perf::us> timer;
  timer.start();
  if (values_hps == nullptr) {
    values_hps = new const uint8_t *[insert_num];
    for (int i = 0; i < insert_num; i++) {
      values_hps[i] = nullptr;
    }
  }
  timer.stop();
  // calculate size to pre-pin

  perf::CpuTimer<perf::us> timer_pin;
  timer_pin.start();
  int keys_hexs_size = util::elements_size_sum(keys_hexs_indexs, insert_num);
  int keys_indexs_size = util::indexs_size_sum(insert_num);
  int64_t values_bytes_size =
      util::elements_size_sum(values_bytes_indexs, insert_num);
  int values_indexs_size = util::indexs_size_sum(insert_num);
  // int values_hps_size = insert_num;

  // printf("keys_hexs_size: %d\n", keys_hexs_size);
  // printf("keys_indexs_size: %d\n", keys_indexs_size);
  // printf("values_bytes_size: %ld\n", values_bytes_size);
  // printf("values_indexs_size: %d\n", values_indexs_size);
  // printf("values_hps_size: %d\n", values_hps_size);
  timer_pin.stop();

  // printf("pre-pin mmeory: %dus\n", timer_pin.get());

  // TODO
  // for 100000: insert is the same w/o pinhost
  // w/ pinhost: load_constants + init is faster
  // w/o pinhost: load_constants + init is slower
  // CHECK_ERROR(gutil::PinHost(keys_hexs, keys_hexs_size));
  // CHECK_ERROR(gutil::PinHost(keys_hexs_indexs, keys_indexs_size));
  // CHECK_ERROR(gutil::PinHost(values_bytes, values_bytes_size));
  // CHECK_ERROR(gutil::PinHost(values_bytes_indexs, values_indexs_size));
  // CHECK_ERROR(gutil::PinHost(values_hps, values_hps_size));
  timer.stop();

  // GPUHashMultiThread::load_constants();
  GpuMPT::Compress::MPT &gpu_mpt_olc = *mpt;
  timer.stop();

  auto [d_hash_nodes, hash_nodes_num] =
      gpu_mpt_olc.puts_latching_with_valuehp_v2(
          keys_hexs, keys_hexs_indexs, values_bytes, values_bytes_indexs,
          values_hps, insert_num);
  gpu_mpt_olc.hash_onepass_v2(d_hash_nodes, hash_nodes_num);

  auto [hash, hash_size] = gpu_mpt_olc.get_root_hash();
  timer.stop();
  // printf("GPU olc hash is: ");
  cutil::println_hex(hash, hash_size);

  // CHECK_ERROR(hipDeviceReset());
  timer.stop();

  // printf(
  //     "\t[Timer] valuehps: %dus\n"
  //     "\t[Timer] pre-pin mmeory: %dus\n"
  //     "\t[Timer] ld constant & init: %dus\n"
  //     "\t[Timer] insert: %dus\n"
  //     "\t[Timer] reset: %dus\n",
  //     timer.get(0), timer.get(1), timer.get(2), timer.get(3), timer.get(4));
  return hash;
}

const uint8_t *build_mpt_olc(enum TrieType trie_type, const uint8_t *keys_hexs,
                             int *keys_hexs_indexs, const uint8_t *values_bytes,
                             int64_t *values_bytes_indexs,
                             const uint8_t **values_hps, int insert_num) {
  CHECK_ERROR(hipSetDevice(1));
  assert(tries != nullptr);
  GpuMPT::Compress::MPT *mpt = nullptr;
  if (trie_type == TrieType::RECEIPT_TRIE) {
    mpt = tries->receipt_trie;
  } else if (trie_type == TrieType::STATE_TRIE) {
    mpt = tries->state_trie;
  } else if (trie_type == TrieType::TRANSACTION_TRIE) {
    mpt = tries->transaction_trie;
  }
  assert(mpt != nullptr);

  // choose the second GPU
  perf::CpuMultiTimer<perf::us> timer;
  timer.start();
  if (values_hps == nullptr) {
    values_hps = new const uint8_t *[insert_num];
    for (int i = 0; i < insert_num; i++) {
      values_hps[i] = nullptr;
    }
  }
  timer.stop();
  // calculate size to pre-pin

  perf::CpuTimer<perf::us> timer_pin;
  timer_pin.start();
  int keys_hexs_size = util::elements_size_sum(keys_hexs_indexs, insert_num);
  int keys_indexs_size = util::indexs_size_sum(insert_num);
  int64_t values_bytes_size =
      util::elements_size_sum(values_bytes_indexs, insert_num);
  int values_indexs_size = util::indexs_size_sum(insert_num);
  // int values_hps_size = insert_num;

  // printf("keys_hexs_size: %d\n", keys_hexs_size);
  // printf("keys_indexs_size: %d\n", keys_indexs_size);
  // printf("values_bytes_size: %ld\n", values_bytes_size);
  // printf("values_indexs_size: %d\n", values_indexs_size);
  // printf("values_hps_size: %d\n", values_hps_size);
  timer_pin.stop();

  // printf("pre-pin mmeory: %dus\n", timer_pin.get());

  // TODO
  // for 100000: insert is the same w/o pinhost
  // w/ pinhost: load_constants + init is faster
  // w/o pinhost: load_constants + init is slower
  // CHECK_ERROR(gutil::PinHost(keys_hexs, keys_hexs_size));
  // CHECK_ERROR(gutil::PinHost(keys_hexs_indexs, keys_indexs_size));
  // CHECK_ERROR(gutil::PinHost(values_bytes, values_bytes_size));
  // CHECK_ERROR(gutil::PinHost(values_bytes_indexs, values_indexs_size));
  // CHECK_ERROR(gutil::PinHost(values_hps, values_hps_size));
  timer.stop();

  // GPUHashMultiThread::load_constants();
  GpuMPT::Compress::MPT &gpu_mpt_olc = *mpt;
  timer.stop();

  auto [d_hash_nodes, hash_nodes_num] =
      gpu_mpt_olc.puts_latching_with_valuehp_v2(
          keys_hexs, keys_hexs_indexs, values_bytes, values_bytes_indexs,
          values_hps, insert_num);
  gpu_mpt_olc.hash_onepass_v2(d_hash_nodes, hash_nodes_num);

  auto [hash, hash_size] = gpu_mpt_olc.get_root_hash();
  timer.stop();
  // printf("GPU olc hash is: ");
  cutil::println_hex(hash, hash_size);

  // CHECK_ERROR(hipDeviceReset());
  timer.stop();

  // printf(
  //     "\t[Timer] valuehps: %dus\n"
  //     "\t[Timer] pre-pin mmeory: %dus\n"
  //     "\t[Timer] ld constant & init: %dus\n"
  //     "\t[Timer] insert: %dus\n"
  //     "\t[Timer] reset: %dus\n",
  //     timer.get(0), timer.get(1), timer.get(2), timer.get(3), timer.get(4));
  return hash;
}

struct nodeset get_all_nodes(enum TrieType trie_type, const uint8_t *keys_hexs,
                             int *keys_hexs_indexs, int num) {
  printf("get_all_nodes\n");
  CHECK_ERROR(hipSetDevice(1));

  assert(tries != nullptr && trie_type == TrieType::STATE_TRIE);
  GpuMPT::Compress::MPT *mpt = tries->state_trie;
  delete tries->receipt_trie;
  delete tries->transaction_trie;
  // if (trie_type == TrieType::RECEIPT_TRIE) {
  //   mpt = tries->receipt_trie;
  // } else if (trie_type == TrieType::STATE_TRIE) {
  //   mpt = tries->state_trie;
  // } else if (trie_type == TrieType::TRANSACTION_TRIE) {
  //   mpt = tries->transaction_trie;
  // }
  assert(mpt != nullptr);
  struct nodeset set {};

  mpt->flush_dirty_nodes(keys_hexs, keys_hexs_indexs, num, set.hashs, set.encs,
                         set.encs_indexs, set.num);
  return set;
}

//
const uint8_t *insert_mpt_2phase(void *mpt, const uint8_t *keys_hexs,
                                 int *keys_hexs_indexs,
                                 const uint8_t *values_bytes,
                                 int64_t *values_bytes_indexs,
                                 const uint8_t **values_hps, int insert_num) {
  assert(mpt != nullptr);

  perf::CpuMultiTimer<perf::us> timer;
  timer.start();
  if (values_hps == nullptr) {
    values_hps = new const uint8_t *[insert_num];
    for (int i = 0; i < insert_num; i++) {
      values_hps[i] = nullptr;
    }
  }
  timer.stop();
  // calculate size to pre-pin

  // perf::CpuTimer<perf::us> timer_pin;
  // timer_pin.start();
  int keys_hexs_size = util::elements_size_sum(keys_hexs_indexs, insert_num);
  int keys_indexs_size = util::indexs_size_sum(insert_num);
  int64_t values_bytes_size =
      util::elements_size_sum(values_bytes_indexs, insert_num);
  int values_indexs_size = util::indexs_size_sum(insert_num);
  // int values_hps_size = insert_num;

  // printf("keys_hexs_size: %d\n", keys_hexs_size);
  // printf("keys_indexs_size: %d\n", keys_indexs_size);
  // printf("values_bytes_size: %ld\n", values_bytes_size);
  // printf("values_indexs_size: %d\n", values_indexs_size);
  // printf("values_hps_size: %d\n", values_hps_size);
  // timer_pin.stop();

  // printf("pre-pin mmeory: %dus\n", timer_pin.get());

  // TODO
  // for 100000: insert is the same w/o pinhost
  // w/ pinhost: load_constants + init is faster
  // w/o pinhost: load_constants + init is slower
  // CHECK_ERROR(gutil::PinHost(keys_hexs, keys_hexs_size));
  // CHECK_ERROR(gutil::PinHost(keys_hexs_indexs, keys_indexs_size));
  // CHECK_ERROR(gutil::PinHost(values_bytes, values_bytes_size));
  // CHECK_ERROR(gutil::PinHost(values_bytes_indexs, values_indexs_size));
  // CHECK_ERROR(gutil::PinHost(values_hps, values_hps_size));
  // timer.stop();

  // GPUHashMultiThread::load_constants();
  ThreadSafeMPT &thread_safe_mpt = *(ThreadSafeMPT *)mpt;

  {
    std::lock_guard<std::mutex> lck(thread_safe_mpt.mu);
    GpuMPT::Compress::MPT &gpu_mpt_olc = thread_safe_mpt.mpt;
    // timer.stop();

    // TODO: free d_keys_hexs_indexs
    // TODO: free d_value_indexs
    auto [d_hash_nodes, hash_nodes_num] = gpu_mpt_olc.puts_2phase_with_valuehp(
        keys_hexs, keys_hexs_indexs, values_bytes, values_bytes_indexs,
        values_hps, insert_num);
    gpu_mpt_olc.hash_onepass_v2(d_hash_nodes, hash_nodes_num);
    auto [hash, hash_size] = gpu_mpt_olc.get_root_hash();

    assert(hash_size == HASH_SIZE);
    memcpy(thread_safe_mpt.hash, hash, hash_size);
    thread_safe_mpt.hash_size = hash_size;

    // timer.stop();
    // printf("GPU olc hash is: ");
    cutil::println_hex(hash, hash_size);

    // CHECK_ERROR(hipDeviceReset());
    // timer.stop();

    // printf(
    //     "\t[Timer] valuehps: %dus\n"
    //     "\t[Timer] pre-pin mmeory: %dus\n"
    //     "\t[Timer] ld constant & init: %dus\n"
    //     "\t[Timer] insert: %dus\n"
    //     "\t[Timer] reset: %dus\n",
    //     timer.get(0), timer.get(1), timer.get(2), timer.get(3),
    //     timer.get(4));
    return hash;
  }
}

const uint8_t *insert_mpt_olc(void *mpt, const uint8_t *keys_hexs,
                              int *keys_hexs_indexs,
                              const uint8_t *values_bytes,
                              int64_t *values_bytes_indexs,
                              const uint8_t **values_hps, int insert_num) {
  assert(mpt != nullptr);

  perf::CpuMultiTimer<perf::us> timer;
  timer.start();
  if (values_hps == nullptr) {
    values_hps = new const uint8_t *[insert_num];
    for (int i = 0; i < insert_num; i++) {
      values_hps[i] = nullptr;
    }
  }
  timer.stop();
  // calculate size to pre-pin

  // perf::CpuTimer<perf::us> timer_pin;
  // timer_pin.start();
  int keys_hexs_size = util::elements_size_sum(keys_hexs_indexs, insert_num);
  int keys_indexs_size = util::indexs_size_sum(insert_num);
  int64_t values_bytes_size =
      util::elements_size_sum(values_bytes_indexs, insert_num);
  int values_indexs_size = util::indexs_size_sum(insert_num);
  // int values_hps_size = insert_num;

  // printf("keys_hexs_size: %d\n", keys_hexs_size);
  // printf("keys_indexs_size: %d\n", keys_indexs_size);
  // printf("values_bytes_size: %ld\n", values_bytes_size);
  // printf("values_indexs_size: %d\n", values_indexs_size);
  // printf("values_hps_size: %d\n", values_hps_size);
  // timer_pin.stop();

  // printf("pre-pin mmeory: %dus\n", timer_pin.get());

  ThreadSafeMPT &thread_safe_mpt = *(ThreadSafeMPT *)mpt;

  {
    std::lock_guard<std::mutex> lck(thread_safe_mpt.mu);
    GpuMPT::Compress::MPT &gpu_mpt_olc = thread_safe_mpt.mpt;

    auto [d_hash_nodes, hash_nodes_num] =
        gpu_mpt_olc.puts_latching_with_valuehp_v2(
            keys_hexs, keys_hexs_indexs, values_bytes, values_bytes_indexs,
            values_hps, insert_num);
    gpu_mpt_olc.hash_onepass_v2(d_hash_nodes, hash_nodes_num);

    auto [hash, hash_size] = gpu_mpt_olc.get_root_hash();

    assert(hash_size == HASH_SIZE);
    memcpy(thread_safe_mpt.hash, hash, hash_size);
    thread_safe_mpt.hash_size = hash_size;
    // timer.stop();
    // printf("GPU olc hash is: ");
    cutil::println_hex(hash, hash_size);

    // CHECK_ERROR(hipDeviceReset());
    // timer.stop();

    // printf(
    //     "\t[Timer] valuehps: %dus\n"
    //     "\t[Timer] pre-pin mmeory: %dus\n"
    //     "\t[Timer] ld constant & init: %dus\n"
    //     "\t[Timer] insert: %dus\n"
    //     "\t[Timer] reset: %dus\n",
    //     timer.get(0), timer.get(1), timer.get(2), timer.get(3),
    //     timer.get(4));
    return hash;
  }
}

///
/// get value and proofs
void get_proofs(  //
    void *mpt, const uint8_t *keys_hexs, int *keys_hexs_indexs, int get_num,
    const uint8_t **&values_hps_get, const int *&values_sizes_get,  // values
    const uint8_t *&proofs, const int *&proofs_indexs, const uint8_t *&hash,
    int &hash_size) {  // proofs
  assert(!values_hps_get && !values_sizes_get && !proofs && !proofs_indexs);
  ThreadSafeMPT &thread_safe_mpt = *(ThreadSafeMPT *)mpt;
  {
    std::lock_guard<std::mutex> lck(thread_safe_mpt.mu);
    GpuMPT::Compress::MPT &gpu_mpt_olc = thread_safe_mpt.mpt;

    // perf::CpuTimer<perf::us> timer;
    // timer.start();

    uint8_t *m_proofs = nullptr;
    int *m_proofs_indexs = nullptr;
    auto m_values_hps_get = new const uint8_t *[get_num];
    auto m_values_sizes_get = new int[get_num];

    gpu_mpt_olc.get_proofs(keys_hexs, keys_hexs_indexs, get_num,
                           m_values_hps_get, m_values_sizes_get, m_proofs,
                           m_proofs_indexs);

    values_hps_get = m_values_hps_get, values_sizes_get = m_values_sizes_get;
    proofs = m_proofs, proofs_indexs = m_proofs_indexs;

    assert(thread_safe_mpt.hash_size == HASH_SIZE);
    uint8_t *hash_ = new uint8_t[thread_safe_mpt.hash_size];
    memcpy(hash_, thread_safe_mpt.hash, thread_safe_mpt.hash_size);

    hash = hash_, hash_size = thread_safe_mpt.hash_size;

    // timer.stop();
    // printf("get_proofs time inside the lock section is %d\n", timer.get());
  }
}

bool verify_proof_single(const uint8_t *key_hex, int key_hex_size,
                         const uint8_t *digest, int digest_size,
                         const uint8_t *value, int value_size,
                         const uint8_t *proof, int proof_size) {
  // TODO
  return GpuMPT::Compress::MPT::verify_proof_cpu(key_hex, key_hex_size, digest,
                                                 digest_size, value, value_size,
                                                 proof, proof_size);
}

void *init_mpt() {
  // assert(tries == nullptr);
  GPUHashMultiThread::load_constants();
  // return new GpuMPT::Compress::MPT{};
  return new ThreadSafeMPT;
}

void free_mpt(void *mpt) {
  // TODO: currently free all
  CHECK_ERROR(hipDeviceReset());
}