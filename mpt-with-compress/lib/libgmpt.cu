// #define GETH 1
#define LEDGERDB 1

#include <unistd.h>

#include <tuple>

#include "libgmpt.h"
#include "mpt/cpu_mpt.cuh"
#include "mpt/gpu_mpt.cuh"
#include "mpt/node.cuh"
#include "util/experiments.cuh"
#include "util/utils.cuh"

struct Tries {
  GpuMPT::Compress::MPT *state_trie;
  GpuMPT::Compress::MPT *transaction_trie;
  GpuMPT::Compress::MPT *receipt_trie;
};

static struct Tries *tries = nullptr;

// TODO: Redefine ALLOCATOR
int preprocess() {
  printf("preprocess()\n");
  CHECK_ERROR(hipSetDevice(1));
  CHECK_ERROR(hipDeviceReset());
  if (tries != nullptr) {
    // delete (GpuMPT::Compress::MPT *)tries->state_trie;
    // delete (GpuMPT::Compress::MPT *)tries->receipt_trie;
    // delete (GpuMPT::Compress::MPT *)tries->transaction_trie;
    delete tries;
  }
  GPUHashMultiThread::load_constants();
  tries = new Tries{};
  tries->state_trie = new GpuMPT::Compress::MPT{};
  tries->receipt_trie = new GpuMPT::Compress::MPT{};
  tries->transaction_trie = new GpuMPT::Compress::MPT{};
  printf("state_trie %p, receipt_trie %p, transaction_trie %p\n",
         tries->state_trie, tries->receipt_trie, tries->transaction_trie);
  return 1;
}

const uint8_t *build_mpt_2phase(enum TrieType trie_type,
                                const uint8_t *keys_hexs, int *keys_hexs_indexs,
                                const uint8_t *values_bytes,
                                int64_t *values_bytes_indexs,
                                const uint8_t **values_hps, int insert_num) {
  CHECK_ERROR(hipSetDevice(1));
  assert(tries != nullptr);
  GpuMPT::Compress::MPT *mpt = nullptr;
  if (trie_type == TrieType::RECEIPT_TRIE) {
    mpt = tries->receipt_trie;
  } else if (trie_type == TrieType::STATE_TRIE) {
    mpt = tries->state_trie;
  } else if (trie_type == TrieType::TRANSACTION_TRIE) {
    mpt = tries->transaction_trie;
  }
  assert(mpt != nullptr);

  // choose the second GPU
  perf::CpuMultiTimer<perf::us> timer;
  timer.start();
  if (values_hps == nullptr) {
    values_hps = new const uint8_t *[insert_num];
    for (int i = 0; i < insert_num; i++) {
      values_hps[i] = nullptr;
    }
  }
  timer.stop();
  // calculate size to pre-pin

  perf::CpuTimer<perf::us> timer_pin;
  timer_pin.start();
  int keys_hexs_size = util::elements_size_sum(keys_hexs_indexs, insert_num);
  int keys_indexs_size = util::indexs_size_sum(insert_num);
  int64_t values_bytes_size =
      util::elements_size_sum(values_bytes_indexs, insert_num);
  int values_indexs_size = util::indexs_size_sum(insert_num);
  // int values_hps_size = insert_num;

  // printf("keys_hexs_size: %d\n", keys_hexs_size);
  // printf("keys_indexs_size: %d\n", keys_indexs_size);
  // printf("values_bytes_size: %ld\n", values_bytes_size);
  // printf("values_indexs_size: %d\n", values_indexs_size);
  // printf("values_hps_size: %d\n", values_hps_size);
  timer_pin.stop();

  // printf("pre-pin mmeory: %dus\n", timer_pin.get());

  // TODO
  // for 100000: insert is the same w/o pinhost
  // w/ pinhost: load_constants + init is faster
  // w/o pinhost: load_constants + init is slower
  // CHECK_ERROR(gutil::PinHost(keys_hexs, keys_hexs_size));
  // CHECK_ERROR(gutil::PinHost(keys_hexs_indexs, keys_indexs_size));
  // CHECK_ERROR(gutil::PinHost(values_bytes, values_bytes_size));
  // CHECK_ERROR(gutil::PinHost(values_bytes_indexs, values_indexs_size));
  // CHECK_ERROR(gutil::PinHost(values_hps, values_hps_size));
  timer.stop();

  // GPUHashMultiThread::load_constants();
  GpuMPT::Compress::MPT &gpu_mpt_olc = *mpt;
  timer.stop();

  auto [d_hash_nodes, hash_nodes_num] =
      gpu_mpt_olc.puts_latching_with_valuehp_v2(
          keys_hexs, keys_hexs_indexs, values_bytes, values_bytes_indexs,
          values_hps, insert_num);
  gpu_mpt_olc.hash_onepass_v2(d_hash_nodes, hash_nodes_num);

  auto [hash, hash_size] = gpu_mpt_olc.get_root_hash();
  timer.stop();
  // printf("GPU olc hash is: ");
  cutil::println_hex(hash, hash_size);

  // CHECK_ERROR(hipDeviceReset());
  timer.stop();

  // printf(
  //     "\t[Timer] valuehps: %dus\n"
  //     "\t[Timer] pre-pin mmeory: %dus\n"
  //     "\t[Timer] ld constant & init: %dus\n"
  //     "\t[Timer] insert: %dus\n"
  //     "\t[Timer] reset: %dus\n",
  //     timer.get(0), timer.get(1), timer.get(2), timer.get(3), timer.get(4));
  return hash;
}

const uint8_t *build_mpt_olc(enum TrieType trie_type, const uint8_t *keys_hexs,
                             int *keys_hexs_indexs, const uint8_t *values_bytes,
                             int64_t *values_bytes_indexs,
                             const uint8_t **values_hps, int insert_num) {
  CHECK_ERROR(hipSetDevice(1));
  assert(tries != nullptr);
  GpuMPT::Compress::MPT *mpt = nullptr;
  if (trie_type == TrieType::RECEIPT_TRIE) {
    mpt = tries->receipt_trie;
  } else if (trie_type == TrieType::STATE_TRIE) {
    mpt = tries->state_trie;
  } else if (trie_type == TrieType::TRANSACTION_TRIE) {
    mpt = tries->transaction_trie;
  }
  assert(mpt != nullptr);

  // choose the second GPU
  perf::CpuMultiTimer<perf::us> timer;
  timer.start();
  if (values_hps == nullptr) {
    values_hps = new const uint8_t *[insert_num];
    for (int i = 0; i < insert_num; i++) {
      values_hps[i] = nullptr;
    }
  }
  timer.stop();
  // calculate size to pre-pin

  perf::CpuTimer<perf::us> timer_pin;
  timer_pin.start();
  int keys_hexs_size = util::elements_size_sum(keys_hexs_indexs, insert_num);
  int keys_indexs_size = util::indexs_size_sum(insert_num);
  int64_t values_bytes_size =
      util::elements_size_sum(values_bytes_indexs, insert_num);
  int values_indexs_size = util::indexs_size_sum(insert_num);
  // int values_hps_size = insert_num;

  // printf("keys_hexs_size: %d\n", keys_hexs_size);
  // printf("keys_indexs_size: %d\n", keys_indexs_size);
  // printf("values_bytes_size: %ld\n", values_bytes_size);
  // printf("values_indexs_size: %d\n", values_indexs_size);
  // printf("values_hps_size: %d\n", values_hps_size);
  timer_pin.stop();

  // printf("pre-pin mmeory: %dus\n", timer_pin.get());

  // TODO
  // for 100000: insert is the same w/o pinhost
  // w/ pinhost: load_constants + init is faster
  // w/o pinhost: load_constants + init is slower
  // CHECK_ERROR(gutil::PinHost(keys_hexs, keys_hexs_size));
  // CHECK_ERROR(gutil::PinHost(keys_hexs_indexs, keys_indexs_size));
  // CHECK_ERROR(gutil::PinHost(values_bytes, values_bytes_size));
  // CHECK_ERROR(gutil::PinHost(values_bytes_indexs, values_indexs_size));
  // CHECK_ERROR(gutil::PinHost(values_hps, values_hps_size));
  timer.stop();

  // GPUHashMultiThread::load_constants();
  GpuMPT::Compress::MPT &gpu_mpt_olc = *mpt;
  timer.stop();

  auto [d_hash_nodes, hash_nodes_num] =
      gpu_mpt_olc.puts_latching_with_valuehp_v2(
          keys_hexs, keys_hexs_indexs, values_bytes, values_bytes_indexs,
          values_hps, insert_num);
  gpu_mpt_olc.hash_onepass_v2(d_hash_nodes, hash_nodes_num);

  auto [hash, hash_size] = gpu_mpt_olc.get_root_hash();
  timer.stop();
  // printf("GPU olc hash is: ");
  cutil::println_hex(hash, hash_size);

  // CHECK_ERROR(hipDeviceReset());
  timer.stop();

  // printf(
  //     "\t[Timer] valuehps: %dus\n"
  //     "\t[Timer] pre-pin mmeory: %dus\n"
  //     "\t[Timer] ld constant & init: %dus\n"
  //     "\t[Timer] insert: %dus\n"
  //     "\t[Timer] reset: %dus\n",
  //     timer.get(0), timer.get(1), timer.get(2), timer.get(3), timer.get(4));
  return hash;
}

struct nodeset get_all_nodes(enum TrieType trie_type, const uint8_t *keys_hexs,
                             int *keys_hexs_indexs, int num) {
  printf("get_all_nodes\n");
  CHECK_ERROR(hipSetDevice(1));

  assert(tries != nullptr && trie_type == TrieType::STATE_TRIE);
  GpuMPT::Compress::MPT *mpt = tries->state_trie;
  delete tries->receipt_trie;
  delete tries->transaction_trie;
  // if (trie_type == TrieType::RECEIPT_TRIE) {
  //   mpt = tries->receipt_trie;
  // } else if (trie_type == TrieType::STATE_TRIE) {
  //   mpt = tries->state_trie;
  // } else if (trie_type == TrieType::TRANSACTION_TRIE) {
  //   mpt = tries->transaction_trie;
  // }
  assert(mpt != nullptr);
  struct nodeset set {};

  mpt->flush_dirty_nodes(keys_hexs, keys_hexs_indexs, num, set.hashs, set.encs,
                         set.encs_indexs, set.num);
  return set;
}

const uint8_t *insert_mpt_2phase(void *mpt, const uint8_t *keys_hexs,
                                 int *keys_hexs_indexs,
                                 const uint8_t *values_bytes,
                                 int64_t *values_bytes_indexs,
                                 const uint8_t **values_hps, int insert_num) {
  // TODO
}

const uint8_t *insert_mpt_olc(void *mpt, const uint8_t *keys_hexs,
                              int *keys_hexs_indexs,
                              const uint8_t *values_bytes,
                              int64_t *values_bytes_indexs,
                              const uint8_t **values_hps, int insert_num) {
  assert(mpt != nullptr);

  perf::CpuMultiTimer<perf::us> timer;
  timer.start();
  if (values_hps == nullptr) {
    values_hps = new const uint8_t *[insert_num];
    for (int i = 0; i < insert_num; i++) {
      values_hps[i] = nullptr;
    }
  }
  timer.stop();
  // calculate size to pre-pin

  // perf::CpuTimer<perf::us> timer_pin;
  // timer_pin.start();
  int keys_hexs_size = util::elements_size_sum(keys_hexs_indexs, insert_num);
  int keys_indexs_size = util::indexs_size_sum(insert_num);
  int64_t values_bytes_size =
      util::elements_size_sum(values_bytes_indexs, insert_num);
  int values_indexs_size = util::indexs_size_sum(insert_num);
  // int values_hps_size = insert_num;

  // printf("keys_hexs_size: %d\n", keys_hexs_size);
  // printf("keys_indexs_size: %d\n", keys_indexs_size);
  // printf("values_bytes_size: %ld\n", values_bytes_size);
  // printf("values_indexs_size: %d\n", values_indexs_size);
  // printf("values_hps_size: %d\n", values_hps_size);
  // timer_pin.stop();

  // printf("pre-pin mmeory: %dus\n", timer_pin.get());

  // TODO
  // for 100000: insert is the same w/o pinhost
  // w/ pinhost: load_constants + init is faster
  // w/o pinhost: load_constants + init is slower
  // CHECK_ERROR(gutil::PinHost(keys_hexs, keys_hexs_size));
  // CHECK_ERROR(gutil::PinHost(keys_hexs_indexs, keys_indexs_size));
  // CHECK_ERROR(gutil::PinHost(values_bytes, values_bytes_size));
  // CHECK_ERROR(gutil::PinHost(values_bytes_indexs, values_indexs_size));
  // CHECK_ERROR(gutil::PinHost(values_hps, values_hps_size));
  timer.stop();

  // GPUHashMultiThread::load_constants();
  GpuMPT::Compress::MPT &gpu_mpt_olc = *(GpuMPT::Compress::MPT *)mpt;
  timer.stop();

  auto [d_hash_nodes, hash_nodes_num] =
      gpu_mpt_olc.puts_latching_with_valuehp_v2(
          keys_hexs, keys_hexs_indexs, values_bytes, values_bytes_indexs,
          values_hps, insert_num);
  gpu_mpt_olc.hash_onepass_v2(d_hash_nodes, hash_nodes_num);

  auto [hash, hash_size] = gpu_mpt_olc.get_root_hash();
  timer.stop();
  // printf("GPU olc hash is: ");
  cutil::println_hex(hash, hash_size);

  // CHECK_ERROR(hipDeviceReset());
  timer.stop();

  // printf(
  //     "\t[Timer] valuehps: %dus\n"
  //     "\t[Timer] pre-pin mmeory: %dus\n"
  //     "\t[Timer] ld constant & init: %dus\n"
  //     "\t[Timer] insert: %dus\n"
  //     "\t[Timer] reset: %dus\n",
  //     timer.get(0), timer.get(1), timer.get(2), timer.get(3), timer.get(4));
  return hash;
}

void *init_mpt() {
  // assert(tries == nullptr);
  GPUHashMultiThread::load_constants();
  return new GpuMPT::Compress::MPT{};
}

void free_mpt(void *mpt) {
  // TODO: currently free all
  CHECK_ERROR(hipDeviceReset());
}