#include "hash/cpu_hash.h"
#include "hash/gpu_hash.cuh"
#include "hash/gpu_hash_kernel.cuh"
#include "util/util.cuh"
#include <stdlib.h>
#include <random>
#include <algorithm>

#define DATA_INPUT_LENGTH 1024

void call_keccak_basic_kernel(const uint8_t *in, uint32_t data_byte_len, uint8_t *out)
{
  uint64_t *d_data;
  uint64_t *out_hash;

  uint32_t input_size64 = data_byte_len / 8 + (data_byte_len % 8 == 0 ? 0 : 1);

  GPUHashMultiThread::load_constants();
  CUDA_SAFE_CALL(hipMalloc(&d_data, input_size64 * sizeof(uint64_t)));
  CUDA_SAFE_CALL(hipMalloc(&out_hash, 4 * sizeof(uint64_t)));
  CUDA_SAFE_CALL(hipMemset(out_hash, 0, 4*sizeof(uint64_t)));
  CUDA_SAFE_CALL(hipMemset(d_data, 0, input_size64));
  CUDA_SAFE_CALL(hipMemcpy((uint8_t *)d_data, in, data_byte_len, hipMemcpyHostToDevice));
  keccak_kernel<<<1, 32>>>(d_data, out_hash, data_byte_len * 8);
  CUDA_SAFE_CALL(hipDeviceSynchronize());
  CUDA_SAFE_CALL(hipMemcpy(out, (uint8_t*)out_hash, HASH_SIZE, hipMemcpyDeviceToHost));
  CUDA_SAFE_CALL(hipFree(d_data));
  CUDA_SAFE_CALL(hipFree(out_hash));
}

void data_gen(const uint8_t *&values_bytes, int *&value_indexs, int n, int turn)
{
  // n = 1 << 16;
  std::random_device rd;
  std::mt19937 g(rd());
  std::uniform_int_distribution dist(0, 1 << 8);
  // generate random values
  const int value_size = DATA_INPUT_LENGTH * turn;
  uint8_t *values = new uint8_t[value_size * n]{};
  for (int i = 0; i < value_size * n; ++i)
  {
    values[i] = dist(g);
  }
  values_bytes = values;
  value_indexs = new int[n]{};
  printf("finish generating values\n");
  for (int i = 0; i < n / 32; i++)
  {
    for (int j = 0; j < 32; ++j)
    {
      value_indexs[i * 32 + j] = value_size * j;
    }
  }
}


int main() {
  uint8_t *hash;
  hash = (uint8_t *)malloc(32 * sizeof(uint8_t));
  memset(hash, 0, 32);
  const uint8_t *input = reinterpret_cast<const uint8_t *>("hgfcghvbjk8291982cisacasioedrxdtcvbnvjghfgkkhvgcfgtdxfghjkbvgcfdtxresxtfyghkjhgvcfdxtcghjklnhbvgcfxdxtrfyghjbvgcfxdtfyguhijkbvgcfxdyguhjkbvgcfxdtfyughjbvgcfxdtryfughjbvgcfdtrftyughjbvgcftdrfyughijbvgcfdrttyughibjvgcfdr5t6y8iuhbjvgcfydr57t6uygibhjvgchfydrft");
  CPUHash::calculate_hash(input, 256, hash);
  util::println_hex(hash, 32);

  memset(hash, 0, 32);

  uint8_t *device_input;
  uint8_t *device_hash;
  CUDA_SAFE_CALL(gutil::DeviceAlloc(device_input, 256));
  CUDA_SAFE_CALL(gutil::DeviceAlloc(device_hash, 32));
  CUDA_SAFE_CALL(gutil::DeviceSet(device_input, 0, 256));
  CUDA_SAFE_CALL(gutil::DeviceSet(device_hash, 0, 32));
  CUDA_SAFE_CALL(gutil::CpyHostToDevice(device_input, input, 256));

  GPUHashSingleThread::load_constants();
  GPUHashSingleThread::test_calculate_hash<<<1, 1>>>(device_input, 256,
                                                     device_hash);
  CUDA_SAFE_CALL(hipDeviceSynchronize());
  CUDA_SAFE_CALL(gutil::CpyDeviceToHost(hash, device_hash, 32));

  util::println_hex(hash, 32);

  memset(hash, 0, 32);
  call_keccak_basic_kernel(input, 256, hash);
  util::println_hex(hash, 32);

  CUDA_SAFE_CALL(gutil::DeviceFree(device_hash));
  CUDA_SAFE_CALL(gutil::DeviceFree(device_input));
  free(hash);
  return 0;
}