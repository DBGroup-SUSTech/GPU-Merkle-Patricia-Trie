#include "mpt/cpu_mpt.h"
#include "mpt/gpu_mpt.cuh"
#include "util/util.cuh"
int main() {
  GPUHashMultiThread::load_constants();
  uint8_t values[32]{0x0b, 0xb4, 0x99, 0x70, 0x47, 0x4b, 0x61, 0x74,
                     0xa7, 0x33, 0x2b, 0xd3, 0xa8, 0xe5, 0x40, 0xea,
                     0x4e, 0xfc, 0x6d, 0xf6, 0xeb, 0x17, 0xf9, 0x08,
                     0x83, 0x0d, 0xbe, 0x3f, 0x38, 0xe0, 0x20, 0xd0};
  // const int n = 1;
  // const uint8_t *keys_bytes = reinterpret_cast<const uint8_t *>(keys);
  // const uint8_t *values_bytes = reinterpret_cast<const uint8_t *>(values);
  // int keys_indexs[]{0, 1};
  // int values_indexs[]{0, 63};

  // CpuMPT cpu_mpt;
  // cpu_mpt.puts(keys_bytes, keys_indexs, values_bytes, values_indexs, n,
  //              DeviceT::CPU);
  // GpuMPT gpu_mpt;
  // gpu_mpt.puts(keys_bytes, keys_indexs, values_bytes, values_indexs, n,
  //              DeviceT::CPU);

  // verify cpu & gpu hash
  // const uint8_t *cpu_hash = nullptr;
  // const uint8_t *gpu_hash = nullptr;
  // cpu_mpt.hash(cpu_hash, DeviceT::CPU);
  // gpu_mpt.hash(gpu_hash, DeviceT::CPU);

  // printf("CPU root hash: 0x");
  // for (int i = 0; i < 32; ++i) {
  //   printf("%02x", cpu_hash[i]);
  // }
  // printf("\n");
  // printf("GPU root hash: 0x");
  // for (int i = 0; i < 32; ++i) {
  //   printf("%02x", gpu_hash[i]);
  // }
  // printf("\n");

  // single hash test
  uint8_t *d_value, *d_hash;
  CHECK_ERROR(gutil::DeviceAlloc(d_value, 32));
  CHECK_ERROR(gutil::CpyHostToDevice(d_value, values, 32));
  CHECK_ERROR(gutil::DeviceAlloc(d_hash, 32));
  CHECK_ERROR(gutil::DeviceSet(d_hash, 0x00, 32));

  // // !! debug
  gkernel::debug::calculate_one_hash<<<1, 32>>>(d_value, 32, d_hash);

  CHECK_ERROR(hipDeviceSynchronize());

  uint8_t h_hash[32]{};
  CHECK_ERROR(gutil::CpyDeviceToHost(h_hash, d_hash, 32));
  printf("data is 0x");
  for (int i = 0; i < 32; ++i) {
    printf("%02x", values[i]);
  }
  printf("\nhash is 0x");
  for (int i = 0; i < 32; ++i) {
    printf("%02x", h_hash[i]);
  }
  printf("\n");

  CHECK_ERROR(gutil::DeviceSet(d_hash, 0x00, 32));

  // call keccak
  keccak_kernel<<<1, 32>>>((uint64_t *)d_value, (uint64_t *)d_hash, 32 * 8);

  CHECK_ERROR(gutil::CpyDeviceToHost(h_hash, d_hash, 32));
  printf("data is 0x");
  for (int i = 0; i < 32; ++i) {
    printf("%02x", values[i]);
  }
  printf("\nhash is 0x");
  for (int i = 0; i < 32; ++i) {
    printf("%02x", h_hash[i]);
  }
  printf("\n");
}