#include "hip/hip_runtime.h"
#include "mpt/cpu_mpt.h"
#include "mpt/gpu_mpt.cuh"
#include "util/timer.h"

#include <iostream>
#include <random>
#include <stdio.h>
#include <string>

__global__ void gpu_addr() {
  printf("GPU address's length = %ld\n", sizeof(Node *));
  printf("size of struct Node is %ld\n", sizeof(Node));
}

void data_gen(const uint8_t *&keys_bytes, int *&keys_indexs,
              const uint8_t *&values_bytes, int *&value_indexs, int &n) {
  n = 1 << 16;
  std::random_device rd;
  std::mt19937 g(rd());
  std::uniform_int_distribution dist(0, 1 << 8);

  // generate keys and shuffle
  uint16_t *keys = new uint16_t[n]{}; // 2 * n byte
  for (int i = 0; i < n; ++i) {
    keys[i] = i;
  }
  std::shuffle(keys, keys + n, g);
  keys_bytes = reinterpret_cast<uint8_t *>(keys);

  printf("finish generating keys\n");

  // generate random values
  const int value_size = 800;
  uint8_t *values = new uint8_t[value_size * n]{};
  for (int i = 0; i < value_size * n; ++i) {
    values[i] = dist(g);
  }
  values_bytes = values;

  printf("finish generating values\n");

  // indexs
  keys_indexs = new int[n * 2]{};
  value_indexs = new int[n * 2]{};
  for (int i = 0; i < n; ++i) {
    keys_indexs[2 * i] = 2 * i;
    keys_indexs[2 * i + 1] = 2 * i + 1;
  }
  for (int i = 0; i < n; ++i) {
    value_indexs[2 * i] = value_size * i;
    value_indexs[2 * i + 1] = value_size * (i + 1) - 1;
  }
}

int main() {
  gpu_addr<<<1, 1>>>();
  hipDeviceSynchronize();

  // prepare data
  // const uint8_t *keys_bytes = reinterpret_cast<const uint8_t
  // *>("helloworld"); int keys_indexs[] = {0, 4, 5, 9}; const uint8_t
  // *values_bytes =
  //     reinterpret_cast<const uint8_t *>("HELLOHELLOWORLDWORLD");
  // int values_indexs[] = {0, 9, 10, 19};

  // input
  const uint8_t *keys_bytes = nullptr;
  int *keys_indexs = nullptr;
  const uint8_t *values_bytes = nullptr;
  int *values_indexs = nullptr;
  int n = 0;
  data_gen(keys_bytes, keys_indexs, values_bytes, values_indexs, n);

  // result
  const uint8_t **values_ptrs = new const uint8_t *[n] {};
  int *values_sizes = new int[n]{};

  std::cout << "finish data gen\n" << std::endl;

  // cpu test puts and gets
  CpuMPT cpu_mpt;
  cpu_mpt.puts(keys_bytes, keys_indexs, values_bytes, values_indexs, n,
               DeviceT::CPU);
  std::fill(values_ptrs, values_ptrs + n, nullptr);
  std::fill(values_sizes, values_sizes + n, 0);

  perf::CpuTimer<perf::us> timer_cpu_get; // timer start ------------
  timer_cpu_get.start();
  cpu_mpt.gets(keys_bytes, keys_indexs, values_ptrs, values_sizes, n,
               DeviceT::CPU);
  timer_cpu_get.stop(); // timer end --------------------------------

  // verify
  // for (int i = 0; i < 2; ++i) {
  //   printf("\nPUT: ");
  //   for (int j = 0; j < element_size(values_indexs, i); ++j) {
  //     printf("%02x", element_start(values_indexs, i, values_bytes)[j]);
  //   }

  //   printf("\nGET: ");
  //   for (int j = 0; j < values_sizes[i]; ++j) {
  //     printf("%02x", values_ptrs[i][j]);
  //   }
  //   printf("\n");
  // }

  // gpu test
  GpuMPT gpu_mpt;
  gpu_mpt.puts(keys_bytes, keys_indexs, values_bytes, values_indexs, n,
               DeviceT::CPU);
  std::fill(values_ptrs, values_ptrs + n, nullptr);
  std::fill(values_sizes, values_sizes + n, 0);

  perf::CpuTimer<perf::ms> timer_gpu_get; // timer start ------------
  timer_gpu_get.start();
  gpu_mpt.gets(keys_bytes, keys_indexs, values_ptrs, values_sizes, n,
               DeviceT::CPU);
  timer_gpu_get.stop(); // timer end --------------------------------

  // verify
  // for (int i = 0; i < 2; ++i) {
  //   printf("\nPUT: ");
  //   for (int j = 0; j < element_size(values_indexs, i); ++j) {
  //     printf("%02x", element_start(values_indexs, i, values_bytes)[j]);
  //   }

  //   printf("\nGET: ");
  //   for (int j = 0; j < values_sizes[i]; ++j) {
  //     printf("%02x", values_ptrs[i][j]);
  //   }
  //   printf("\n");
  // }

  printf("CPU get: %d ms\nGPU get: %d ms\n", timer_cpu_get.get(),
         timer_gpu_get.get());
}
